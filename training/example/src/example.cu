// example.cu

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c, int N) {
    // グローバルスレッドIDを計算
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {  // 範囲チェック
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    const int N = 1024;  // 要素数
    
    size_t bytes = N * sizeof(int);
    hipSetDevice(0);

    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed! " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // ホストメモリの確保
    int *h_a, *h_b, *h_c;
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    // データの初期化
    for (int i = 0; i < N; ++i) {
        h_a[i] = -i;
        h_b[i] = i * i;
    }

    // デバイスメモリの確保
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // ホストからデバイスへのデータ転送
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // カーネルの実行
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // エラーチェック
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // デバイスからホストへのデータ転送
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // 結果の確認
    for (int i = 0; i < N; ++i) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            std::cerr << "Error at index " << i << ": expected " << h_a[i] + h_b[i] << ", got " << h_c[i] << std::endl;
            return 1;
        }
    }

    // メモリの解放
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    std::cout << "All tests passed!" << std::endl;

    return 0;
}