// example.cu

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c, int N) {
    // �O���[�o���X���b�hID���v�Z
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {  // �͈̓`�F�b�N
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    const int N = 1024;  // �v�f��
    
    size_t bytes = N * sizeof(int);
    hipSetDevice(0);

    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed! " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // �z�X�g�������̊m��
    int *h_a, *h_b, *h_c;
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    // �f�[�^�̏�����
    for (int i = 0; i < N; ++i) {
        h_a[i] = -i;
        h_b[i] = i * i;
    }

    // �f�o�C�X�������̊m��
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // �z�X�g����f�o�C�X�ւ̃f�[�^�]��
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // �J�[�l���̎��s
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // �G���[�`�F�b�N
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // �f�o�C�X����z�X�g�ւ̃f�[�^�]��
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // ���ʂ̊m�F
    for (int i = 0; i < N; ++i) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            std::cerr << "Error at index " << i << ": expected " << h_a[i] + h_b[i] << ", got " << h_c[i] << std::endl;
            return 1;
        }
    }

    // �������̉��
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    std::cout << "All tests passed!" << std::endl;

    return 0;
}