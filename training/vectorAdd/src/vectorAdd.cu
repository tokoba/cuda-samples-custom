#include <stdio.h>

#include "hip/hip_runtime.h"


#include "malloc.h"

#define SIZE (2048) /* Define the size of the vectors */

#define TPW (32) /* Threads per Warp (FIXED) */

typedef enum tag_GPU_PRODUCTS {
    RTX4090,
    RTX4080,
    RTX3090,
    MAX_GPU_PRODUCTS
} GPU_PRODUCTS;

/* SM is Streaming Multiprocessor */
/* GPU Hierarcy
 * Grid == GPU itself (all the computational resources)
 * Block == SM - Streaming Multiprocessor (each calculation block)
 * Thread == Warp - 32 threads
 * Core == INT32, FP32 etc(each ALU)
 *
 */
typedef struct tag_GPU_SPEC {
    int cudaCoresPerGrid;   /* CUDA Cores per Grid */
    int tensorCoresPerGrid; /* Tensor Cores per Grid */
    int blockPerGrid;       /* Block per Grid (SM per Grid) */
    int cudaCoresPerSM;     /* CUDA Cores per SM */
    int tensorCoresPerSM;   /* Tensor Cores per SM */
} GPU_SPEC;                 /* Define the GPU Spec */

GPU_SPEC gpu_spec[MAX_GPU_PRODUCTS] = {
    {
        /* RTX4090 (AD102) */
        16384, /* CUDA Core */
        512,   /* Tensor Core */
        128,   /* Block per Grid (SM per Grid) */
        128,   /* CUDA Cores per SM */
        4      /* Tensor Cores per SM */

    },
    {
        /* RTX4080 (AD103) */
        9728, /* CUDA Core */
        304,  /* Tensor Core */
        76,   /* Block per Grid (SM per Grid) */
        128,  /* CUDA Cores per SM */
        4     /* Tensor Cores per SM */
    },
    {
        /* RTX3090 (GB102) */
        10496, /* CUDA Core */
        328,   /* Tensor Core */
        82,    /* Block per Grid (SM per Grid) */
        128,   /* CUDA Cores per SM */
        4      /* Tensor Cores per SM */
    }

}; /* Define the GPU Spec */

/* CUDA Kernel function for vector addition */
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the index is within bounds
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    /* Allocate memory space */
    int *a, *b, *c;        // Host pointers
    int *d_a, *d_b, *d_c;  // Device pointers
    int size = SIZE * sizeof(int);
    hipEvent_t start, stop;
    hipEventCreateWithFlags(&start, 0);
    hipEventCreateWithFlags(&stop, 0);

    hipError_t cudaStatus = hipSuccess;
    enum {
        E_OK = 0,
        E_FAIL = 1
    };

    // Allocate memory on host and initialize
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    /* Allocate device vectors */
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    /* initialize the inputs */
    for (int i = 0; i < SIZE; i++) {
        a[i] = i;
        b[i] = SIZE - i;
    }
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    /* Launch the Vector Add Cuda Kernel 2 blocks*/
    hipEventRecord(start);
    // vectorAdd<<<2, 1024>>>(d_a, d_b, d_c, SIZE);
    vectorAdd<<<128, 16>>>(d_a, d_b, d_c, SIZE);
    hipEventRecord(stop);

    /* Copy result back to host */
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    /* wait for kernel completion */
    cudaStatus = hipDeviceSynchronize();
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time: %f msec\n", milliseconds);
    if (hipSuccess != cudaStatus) {
        printf("test02 cudaStatus Failure, hipDeviceSynchronize failed.\n");
        return E_FAIL;
    }
    printf("Execution finished.\n");
/* Verify the result */
#if 0
    for (int i = 0; i < SIZE; i++) {
        printf("index %d: %d + %d expected %d, got %d\n", i, a[i], b[i], a[i] + b[i], c[i]);
    }
#endif
    /* Free device global memory */
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    /* Free host memory */
    free(a);
    free(b);
    free(c);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Memory freed.\n");

    return 0;
}